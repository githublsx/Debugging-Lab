#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""

//NVTX Dir: C:\Program Files\NVIDIA GPU Computing Toolkit\nvToolsExt
#include <nvToolsExt.h>

//Initialize sizes
const int sizeX = 4096;
const int sizeY = 4096;

using namespace std;

struct DIMS
{
    dim3 dimBlock;
    dim3 dimGrid;
};

#define CUDA(call) do {                             \
    hipError_t e = (call);                         \
    if (e == hipSuccess) break;                    \
    fprintf(stderr, __FILE__":%d: %s (%d)\n",       \
            __LINE__, hipGetErrorString(e), e);    \
    exit(1);                                        \
} while (0)

inline unsigned divup(unsigned n, unsigned div)
{
    return (n + div - 1) / div;
}

// Check errors
void postprocess(const float *ref, const float *res, int n)
{
    bool passed = true;
    for (int i = 0; i < n; i++)
    {
        if (res[i] != ref[i])
        {
            printf("ID:%d \t Res:%f \t Ref:%f\n", i, res[i], ref[i]);
            printf("%25s\n", "*** FAILED ***");
            passed = false;
            break;
        }
    }
    if(passed)
        printf("Post process check passed!!\n");
}

void preprocess(float *res, float *dev_res, int n)
{
    std::fill(res, res + n, -1);
    hipMemset(dev_res, -1, n * sizeof(float));
}

__global__ void copyKernel(const float* const a, float* const b)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;          // Compute correctly - Global X index
    int j = threadIdx.y + blockDim.y * blockIdx.y;          // Compute correctly - Global Y index

    int index = i + sizeX * j;      // Compute 1D index from i, j

    b[index] = a[index];
}

__global__ void matrixTransposeNaive(const float* const a, float* const b)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;          // Compute correctly - Global X index
    int j = threadIdx.y + blockDim.y * blockIdx.y;          // Compute correctly - Global Y index

    int index_in  = i + sizeX * j;  // Compute input index (i,j) from matrix A
    int index_out = i * sizeY + j;  // Compute output index (j,i) in matrix B = transpose(A)

    // Copy data from A to B
    b[index_out] = a[index_in];
}

template<int BLOCK_SIZE_X, int BLOCK_SIZE_Y>
__global__ void matrixTransposeShared(const float* const a, float* const b)
{
    //Allocate appropriate shared memory
    __shared__ float mat[BLOCK_SIZE_Y][BLOCK_SIZE_X];

    //Compute input and output index
    int bx = blockIdx.x * blockDim.x;     // Compute block offset - this is number of global threads in X before this block
    int by = blockIdx.y * blockDim.y;     // Compute block offset - this is number of global threads in Y before this block
    int i  = threadIdx.x + bx;     // Global input x index - Same as previous kernels
    int j  = threadIdx.y + by;     // Global input y index - Same as previous kernels
    int ti = threadIdx.x + by;     // Global output x index - remember the transpose
    int tj = threadIdx.y + bx;     // Global output y index - remember the transpose

    //Copy data from input to shared memory
	mat[threadIdx.y][threadIdx.x] = a[i + sizeX * j];

	__syncthreads();

    //Copy data from shared memory to global memory b
	b[tj * sizeX + ti] = mat[threadIdx.x][threadIdx.y];
}

__global__ void matrixTransposeDynamicShared(const float* const a, float* const b)
{
    //Allocate appropriate shared memory
    extern __shared__ float mat[];

    //Compute input and output index - same as matrixTransposeShared kernel
	int bx = blockIdx.x * blockDim.x;     // Compute block offset - this is number of global threads in X before this block
	int by = blockIdx.y * blockDim.y;     // Compute block offset - this is number of global threads in Y before this block
	int i = threadIdx.x + bx;     // Global input x index - Same as previous kernels
	int j = threadIdx.y + by;     // Global input y index - Same as previous kernels
	int ti = threadIdx.x + by;     // Global output x index - remember the transpose
	int tj = threadIdx.y + bx;     // Global output y index - remember the transpose

	//Copy data from input to shared memory
	mat[threadIdx.y * blockDim.x + threadIdx.x] = a[i + sizeX * j];

	__syncthreads();

	//Copy data from shared memory to global memory b
	b[tj * sizeX + ti] = mat[threadIdx.x * blockDim.x + threadIdx.y];
}

int main(int argc, char *argv[])
{
    // Host arrays.
    float* a      = new float[sizeX * sizeY];
    float* b      = new float[sizeX * sizeY];
    float* a_gold = new float[sizeX * sizeY];
    float* b_gold = new float[sizeX * sizeY];

    // Device arrays
    float *d_a, *d_b;

    // Allocate memory on the device
    CUDA(hipMalloc((void **) &d_a, sizeX * sizeY * sizeof(float)));

    CUDA(hipMalloc((void **) &d_b, sizeX * sizeY * sizeof(float)));

    // Fill matrix A
    for (int i = 0; i < sizeX * sizeY; i++)
        a[i] = (float)i;

    cout << endl;

    // Copy array contents of A from the host (CPU) to the device (GPU)
    hipMemcpy(d_a, a, sizeX * sizeY * sizeof(float), hipMemcpyHostToDevice);

    //Compute "gold" reference standard
    for(int jj = 0; jj < sizeY; jj++)
    {
        for(int ii = 0; ii < sizeX; ii++)
        {
            a_gold[jj * sizeX + ii] = a[jj * sizeX + ii];
            b_gold[ii * sizeY + jj] = a[jj * sizeX + ii];
        }
    }

    hipDeviceSynchronize();

#define CPU_TRANSPOSE
#ifdef CPU_TRANSPOSE
    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***CPU Transpose***" << endl;
    {
        for (int jj = 0; jj < sizeY; jj++)
            for (int ii = 0; ii < sizeX; ii++)
                b[ii * sizeX + jj] = a[jj * sizeX + ii];
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////
#endif

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Device To Device Copy***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"

        DIMS dims;
        dims.dimBlock = dim3(16, 16, 1);
        dims.dimGrid  = dim3(sizeX / dims.dimBlock.x,
                             sizeY / dims.dimBlock.y,
                             1);

        // Launch the GPU kernel
        copyKernel<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, sizeX * sizeY * sizeof(float), hipMemcpyDeviceToHost);

        postprocess(a_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Naive Transpose***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        // HINT: Look above for copy kernel dims computation
        DIMS dims;
        dims.dimBlock = dim3(16, 16, 1);
        dims.dimGrid  = dim3(sizeX / dims.dimBlock.x,
                             sizeY / dims.dimBlock.y,
                             1);

        // Launch the GPU kernel
        matrixTransposeNaive<<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, sizeX * sizeY * sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Shared Memory Transpose***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(16, 16, 1);
        dims.dimGrid  = dim3(sizeX / dims.dimBlock.x,
                             sizeY / dims.dimBlock.y,
                             1);

        // Launch the GPU kernel
        matrixTransposeShared<16, 16><<<dims.dimGrid, dims.dimBlock>>>(d_a, d_b);

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, sizeX * sizeY * sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////
    cout << "******************************************" << endl;
    cout << "***Shared Memory Transpose with Dynamic Shared Memory***" << endl;
    {
        preprocess(b, d_b, sizeX * sizeY);
        // Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
        // Calculate number of blocks along X and Y in a 2D CUDA "grid"
        DIMS dims;
        dims.dimBlock = dim3(16, 16, 1);
        dims.dimGrid  = dim3(sizeX / dims.dimBlock.x,
                             sizeY / dims.dimBlock.y,
                             1);

        // Launch the GPU kernel
        int sharedMemoryPerBlockInBytes = dims.dimBlock.x * dims.dimBlock.y * sizeof(float); // Compute This
        matrixTransposeDynamicShared<<<dims.dimGrid, dims.dimBlock, sharedMemoryPerBlockInBytes>>>(d_a, d_b);
		// Call kernel - 

        // copy the answer back to the host (CPU) from the device (GPU)
        hipMemcpy(b, d_b, sizeX * sizeY * sizeof(float), hipMemcpyDeviceToHost);

        postprocess(b_gold, b, sizeX * sizeY);
    }
    cout << "******************************************" << endl;
    cout << endl;
    ////////////////////////////////////////////////////////////

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    delete[] a;
    delete[] b;

    //CUDA Reset for NVProf
    CUDA(hipDeviceReset());

    // successful program termination
    return 0;
}
